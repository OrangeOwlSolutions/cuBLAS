#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <iostream>
#include <iomanip>
#include <hipblas.h>
#include <conio.h>
#include <assert.h>

/**********************/
/* cuBLAS ERROR CHECK */
/**********************/
#ifndef cublasSafeCall
#define cublasSafeCall(err)     __cublasSafeCall(err, __FILE__, __LINE__)
#endif

inline void __cublasSafeCall(hipblasStatus_t err, const char *file, const int line)
{
    if( HIPBLAS_STATUS_SUCCESS != err) {
        fprintf(stderr, "CUBLAS error in file '%s', line %d\n \nerror %d \nterminating!\n",__FILE__, __LINE__,err); 
        getch(); hipDeviceReset(); assert(0); 
    }
}

// convert a linear index to a linear index in the transpose 
struct transpose_index : public thrust::unary_function<size_t,size_t>
{
    size_t m, n;

    __host__ __device__
    transpose_index(size_t _m, size_t _n) : m(_m), n(_n) {}

    __host__ __device__
    size_t operator()(size_t linear_index)
    {
        size_t i = linear_index / n;
        size_t j = linear_index % n;

        return m * j + i;
    }
};

// convert a linear index to a row index
struct row_index : public thrust::unary_function<size_t,size_t>
{
    size_t n;

    __host__ __device__
    row_index(size_t _n) : n(_n) {}

    __host__ __device__

    size_t operator()(size_t i)
    {
        return i / n;
    }
};

// transpose an M-by-N array
template <typename T>
void transpose(size_t m, size_t n, thrust::device_vector<T>& src, thrust::device_vector<T>& dst)
{
    thrust::counting_iterator<size_t> indices(0);

    thrust::gather
    (thrust::make_transform_iterator(indices, transpose_index(n, m)),
    thrust::make_transform_iterator(indices, transpose_index(n, m)) + dst.size(),
    src.begin(),dst.begin());
}

// print an M-by-N array
template <typename T>
void print(size_t m, size_t n, thrust::device_vector<T>& d_data)
{
    thrust::host_vector<T> h_data = d_data;

    for(size_t i = 0; i < m; i++)
    {
        for(size_t j = 0; j < n; j++)
            std::cout << std::setw(8) << h_data[i * n + j] << " ";
            std::cout << "\n";
    }
}

int main(void)
{
    size_t m = 5; // number of rows
    size_t n = 4; // number of columns

    // 2d array stored in row-major order [(0,0), (0,1), (0,2) ... ]
    thrust::device_vector<double> data(m * n, 1.);
    data[1] = 2.;
    data[3] = 3.;

    std::cout << "Initial array" << std::endl;
    print(m, n, data);

    std::cout << "Transpose array - Thrust" << std::endl;
    thrust::device_vector<double> transposed_thrust(m * n);
    transpose(m, n, data, transposed_thrust);
    print(n, m, transposed_thrust);

    std::cout << "Transpose array - cuBLAS" << std::endl;
    thrust::device_vector<double> transposed_cuBLAS(m * n);
    double* dv_ptr_in  = thrust::raw_pointer_cast(data.data());
    double* dv_ptr_out = thrust::raw_pointer_cast(transposed_cuBLAS.data());
    double alpha = 1.;
    double beta  = 0.;
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));
    cublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, dv_ptr_in, n, &beta, dv_ptr_in, n, dv_ptr_out, m)); 
    print(n, m, transposed_cuBLAS);

    getch();

    return 0;
}
