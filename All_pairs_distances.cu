#include "hip/hip_runtime.h"
include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"
#include "TimingGPU.cuh"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

/***********************************************************/
/* SQUARED ABSOLUTE VALUE FUNCTOR - NEEDED FOR APPROACH #1 */
/***********************************************************/
struct abs2 {
	__host__ __device__ double operator()(const float &x) const { return x * x; }
};

// --- Required for approach #2
__device__ float *vals;

/******************************************/
/* ROW_REDUCTION - NEEDED FOR APPROACH #2 */
/******************************************/
struct row_reduction {

    const int Ncols;    // --- Number of columns

    row_reduction(int _Ncols) : Ncols(_Ncols) {}

    __device__ float operator()(float& x, int& y ) {
        float temp = 0.f;
        for (int i = 0; i<Ncols; i++)
            temp += vals[i + (y*Ncols)] * vals[i + (y*Ncols)];
        return temp;
    }
};

/************************************************/
/* KERNEL FUNCTION TO ASSEMBLE THE FINAL RESULT */
/************************************************/
__global__ void assemble_final_result(const float * __restrict__ d_norms_x_2, const float * __restrict__ d_norms_y_2, float * __restrict__ d_dots,
									  const int NX, const int NY) {

	const int i = threadIdx.x + blockIdx.x * gridDim.x;
	const int j = threadIdx.y + blockIdx.y * gridDim.y;

	if ((i < NY) && (j < NX)) d_dots[i * NX+ j] = d_norms_x_2[j] + d_norms_y_2[i] - 2 * d_dots[i * NX+ j];

}

/********/
/* MAIN */
/********/
int main()
{
    //const int Ndims = 128;		// --- Number of rows
    //const int NX	= 1000;		// --- Number of columns
    //const int NY	= 2000;		// --- Number of columns

    const int Ndims = 3;		// --- Number of rows
    const int NX	= 4;		// --- Number of columns
    const int NY	= 5;		// --- Number of columns

	// --- Random uniform integer distribution between 10 and 99
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int> dist(10, 99);

    // --- Matrices allocation and initialization
    thrust::device_vector<float> d_X(Ndims * NX);
    thrust::device_vector<float> d_Y(Ndims * NY);
    for (size_t i = 0; i < d_X.size(); i++) d_X[i] = (float)dist(rng);
    for (size_t i = 0; i < d_Y.size(); i++) d_Y[i] = (float)dist(rng);

    TimingGPU timerGPU;

	// --- cuBLAS handle creation
	hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

	/**********************************************/
    /* CALCULATING THE NORMS OF THE ELEMENTS OF X */
    /**********************************************/
    thrust::device_vector<float> d_norms_x_2(NX);

	// --- Approach nr. 1
	//timerGPU.StartCounter();
	thrust::device_vector<float> d_X_2(Ndims * NX);
	thrust::transform(d_X.begin(), d_X.end(), d_X_2.begin(), abs2());

	thrust::device_vector<float> d_ones(Ndims, 1.f);

    float alpha = 1.f;
    float beta  = 0.f;
    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_T, Ndims, NX, &alpha, thrust::raw_pointer_cast(d_X_2.data()), Ndims, 
                               thrust::raw_pointer_cast(d_ones.data()), 1, &beta, thrust::raw_pointer_cast(d_norms_x_2.data()), 1));
	
	//printf("Timing for approach #1 = %f\n", timerGPU.GetCounter());

    // --- Approach nr. 2
	//timerGPU.StartCounter();
 //   float *s_vals = thrust::raw_pointer_cast(&d_X[0]);
 //   gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(vals), &s_vals, sizeof(float *)));
 //   thrust::transform(d_norms_x_2.begin(), d_norms_x_2.end(), thrust::counting_iterator<int>(0),  d_norms_x_2.begin(), row_reduction(Ndims));

	//printf("Timing for approach #2 = %f\n", timerGPU.GetCounter());

	/**********************************************/
    /* CALCULATING THE NORMS OF THE ELEMENTS OF Y */
    /**********************************************/
    thrust::device_vector<float> d_norms_y_2(NX);

	thrust::device_vector<float> d_Y_2(Ndims * NX);
	thrust::transform(d_Y.begin(), d_Y.end(), d_Y_2.begin(), abs2());

    cublasSafeCall(hipblasSgemv(handle, HIPBLAS_OP_T, Ndims, NY, &alpha, thrust::raw_pointer_cast(d_Y_2.data()), Ndims, 
                               thrust::raw_pointer_cast(d_ones.data()), 1, &beta, thrust::raw_pointer_cast(d_norms_y_2.data()), 1));


	/***********************************/
    /* CALCULATING THE SCALAR PRODUCTS */
    /***********************************/
    thrust::device_vector<float> d_dots(NX * NY);

	cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, NX, NY, Ndims, &alpha,
		                       thrust::raw_pointer_cast(d_X.data()), Ndims, thrust::raw_pointer_cast(d_Y.data()), Ndims, &beta,
							   thrust::raw_pointer_cast(d_dots.data()), NX));

	/*****************************/
	/* ASSEMBLE THE FINAL RESULT */
	/*****************************/
	
	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 dimGrid(iDivUp(NX, BLOCK_SIZE_X), iDivUp(NY, BLOCK_SIZE_Y));
	assemble_final_result<<<dimGrid, dimBlock>>>(thrust::raw_pointer_cast(d_norms_x_2.data()), thrust::raw_pointer_cast(d_norms_y_2.data()), 
		                                         thrust::raw_pointer_cast(d_dots.data()), NX, NY);
	
	for(int i = 0; i < NX * NY; i++) std::cout << d_dots[i] << "\n";

	return 0;
}
