#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/random.h>

#include <hipblas.h>

#include "Utilities.cuh"

/********/
/* MAIN */
/********/
int main()
{
	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
  
	//const int Nrows1 = 10;			// --- Number of rows of matrix 1
	//const int Ncols1 = 10;			// --- Number of columns of matrix 1

	//const int Nrows2 = 15;			// --- Number of rows of matrix 2
	//const int Ncols2 = 15;			// --- Number of columns of matrix 2

	//const int Nrows3 = 12;			// --- Number of rows of matrix 3
	//const int Ncols3 = 12;			// --- Number of columns of matrix 3

	const int Nrows1 = 10;			// --- Number of rows of matrix 1
	const int Ncols1 = 9;			// --- Number of columns of matrix 1

	const int Nrows2 = 15;			// --- Number of rows of matrix 2
	const int Ncols2 = 13;			// --- Number of columns of matrix 2

	const int Nrows3 = 10;			// --- Number of rows of matrix 3
	const int Ncols3 = 12;			// --- Number of columns of matrix 3

	const int Nrows = 5;			// --- Number of rows of submatrix matrix 3 = Number of rows of submatrix 1
	const int Ncols = 3;			// --- Number of columns of submatrix matrix 3 = Number of columns of submatrix 2

	const int Nrowscols = 4;		// --- Number of columns of submatrix 1 and of rows of submatrix 2

	const int x1 = 3;				// --- Offset for submatrix multiplication along the rows
	const int y1 = 2;				// --- Offset for submatrix multiplication along the columns
	
	const int x2 = 6;				// --- Offset for submatrix multiplication along the rows
	const int y2 = 4;				// --- Offset for submatrix multiplication along the columns

	const int x3 = 3;				// --- Offset for submatrix multiplication along the rows
	const int y3 = 5;				// --- Offset for submatrix multiplication along the columns

	// --- Random uniform integer distribution between 0 and 100
	thrust::default_random_engine rng;
	thrust::uniform_int_distribution<int> dist(0, 20);

	// --- Matrix allocation and initialization
	thrust::device_vector<float> d_matrix1(Nrows1 * Ncols1);
	thrust::device_vector<float> d_matrix2(Nrows2 * Ncols2);
	for (size_t i = 0; i < d_matrix1.size(); i++) d_matrix1[i] = (float)dist(rng);
	for (size_t i = 0; i < d_matrix2.size(); i++) d_matrix2[i] = (float)dist(rng);

	printf("\n\nOriginal full size matrix A\n");
	for(int i = 0; i < Nrows1; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols1; j++) 
			std::cout << d_matrix1[j * Nrows1 + i] << " ";
		std::cout << "]\n";
	}

	printf("\n\nOriginal full size matrix B\n");
	for(int i = 0; i < Nrows2; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols2; j++) 
			std::cout << d_matrix2[j * Nrows2 + i] << " ";
		std::cout << "]\n";
	}

	/*************************/
	/* MATRIX MULTIPLICATION */
	/*************************/
	hipblasHandle_t handle;

	cublasSafeCall(hipblasCreate(&handle));

	thrust::device_vector<float> d_matrix3(Nrows3 * Ncols3, 10.f);

	float alpha = 1.f;
	float beta  = 0.f;
	cublasSafeCall(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Nrows, Ncols, Nrowscols, &alpha,
				   thrust::raw_pointer_cast(d_matrix1.data())+x1+Nrows1*y1, Nrows1, thrust::raw_pointer_cast(d_matrix2.data())+x2+Nrows2*y2, Nrows2,
				   &beta, thrust::raw_pointer_cast(d_matrix3.data())+x3+Nrows3*y3, Nrows3));

	printf("\n\nResult full size matrix C\n");
	for(int i = 0; i < Nrows3; i++) {
		std::cout << "[ ";
		for(int j = 0; j < Ncols3; j++) 
			std::cout << d_matrix3[j * Nrows3 + i] << " ";
		std::cout << "]\n";
	}

	return 0; 
}
