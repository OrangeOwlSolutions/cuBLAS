#include <thrust/device_vector.h>
#include <thrust/random.h>

#include <stdio.h>
#include <iostream>

#include "Utilities.cuh"

/********/
/* MAIN */
/********/
int main()
{
    const int N_basis_functions = 5;     // --- Number of rows					-> Number of basis functions
    const int N_sampling_points = 8;     // --- Number of columns				-> Number of sampling points of the basis functions

    // --- Random uniform integer distribution between 10 and 99
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int> dist(10, 99);

    // --- Matrix allocation and initialization
    thrust::device_vector<float> d_basis_functions_real(N_basis_functions * N_sampling_points);
    for (size_t i = 0; i < d_basis_functions_real.size(); i++) d_basis_functions_real[i] = (float)dist(rng);

    thrust::device_vector<double> d_basis_functions_double_real(N_basis_functions * N_sampling_points);
    for (size_t i = 0; i < d_basis_functions_double_real.size(); i++) d_basis_functions_double_real[i] = (double)dist(rng);

	/************************************/
    /* COMPUTING THE LINEAR COMBINATION */
    /************************************/
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

    thrust::device_vector<float>  d_linear_combination_real(N_sampling_points);
    thrust::device_vector<double> d_linear_combination_double_real(N_sampling_points);
    thrust::device_vector<float>  d_coeff_real(N_basis_functions, 1.f);
    thrust::device_vector<double> d_coeff_double_real(N_basis_functions, 1.);

	linearCombination(thrust::raw_pointer_cast(d_coeff_real.data()), thrust::raw_pointer_cast(d_basis_functions_real.data()), thrust::raw_pointer_cast(d_linear_combination_real.data()),
	                  N_basis_functions, N_sampling_points, handle);
	linearCombination(thrust::raw_pointer_cast(d_coeff_double_real.data()), thrust::raw_pointer_cast(d_basis_functions_double_real.data()), thrust::raw_pointer_cast(d_linear_combination_double_real.data()),
	                  N_basis_functions, N_sampling_points, handle);
					   
	/*************************/
    /* DISPLAYING THE RESULT */
    /*************************/
    std::cout << "Real case \n\n";
	for(int j = 0; j < N_sampling_points; j++) {
        std::cout << "Column " << j << " - [ ";
        for(int i = 0; i < N_basis_functions; i++)
            std::cout << d_basis_functions_real[i * N_sampling_points + j] << " ";
        std::cout << "] = " << d_linear_combination_real[j] << "\n";
    }

    std::cout << "\n\nDouble real case \n\n";
	for(int j = 0; j < N_sampling_points; j++) {
        std::cout << "Column " << j << " - [ ";
        for(int i = 0; i < N_basis_functions; i++)
            std::cout << d_basis_functions_double_real[i * N_sampling_points + j] << " ";
        std::cout << "] = " << d_linear_combination_double_real[j] << "\n";
    }

	return 0;
}
